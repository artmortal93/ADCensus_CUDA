#include "adcensus_stereo.h"
#include "scanline_optimize.cuh"
#include "aggregation_util.cuh"
#include "ad_util.cuh"
#include "multistep_refine.cuh"



ADCensusStereo::~ADCensusStereo()
{
	FreeCudaResource();
}

void ADCensusStereo::CleanUpMemory()
{   
	if (!first_time) {
		//only clean up the memory part and transfer memory
		int disparity_min = m_option.min_disparity;
		int disparity_max = m_option.max_disparity;
		int disparity_range = disparity_max - disparity_min;
		//set memory to 0 (if neccessary,seems not..)
		gpuErrchk(hipMemset(census_left_d, 0, width * height * sizeof(uint64_t)));
		gpuErrchk(hipMemset(census_right_d, 0, width * height * sizeof(uint64_t)));
		gpuErrchk(hipMemset(cost_aggr_d, 0, width * height * disparity_range * sizeof(float)));
		gpuErrchk(hipMemset(cost_init_d, 0, width * height * disparity_range * sizeof(float)));
		gpuErrchk(hipMemset(vec_counter_horizontal_d, 0, width * height * sizeof(uint16_t)));
		gpuErrchk(hipMemset(vec_counter_vertical_d, 0, width * height * sizeof(uint16_t)));
		gpuErrchk(hipMemset(vec_counter_buffer_d, 0, width * height * sizeof(uint16_t)));
		gpuErrchk(hipMemset(disp_mask_d, 0, width * height * sizeof(uint8_t)));
		gpuErrchk(hipMemset(disp_left_d, 0, width * height * sizeof(float)));
		gpuErrchk(hipMemset(disp_left_buffer_d, 0, width * height * sizeof(float)));
		gpuErrchk(hipMemset(disp_right_d, 0, width * height * sizeof(float)));
		
	}
}

void ADCensusStereo::AllocateCudaResource()
{
	int disparity_min = m_option.min_disparity;
	int disparity_max = m_option.max_disparity;
	int disparity_range = disparity_max - disparity_min;

	gpuErrchk(hipMalloc((void**)&img_left_rgb_d, width * height * 3 * sizeof(uint8_t)));
	gpuErrchk(hipMalloc((void**)&img_right_rgb_d, width * height * 3 * sizeof(uint8_t)));
	gpuErrchk(hipMalloc((void**)&img_left_gray_d, width * height * sizeof(uint8_t)));
	gpuErrchk(hipMalloc((void**)&img_right_gray_d, width * height * sizeof(uint8_t)));
	gpuErrchk(hipMalloc((void**)&census_left_d, width * height * sizeof(uint64_t)));
	gpuErrchk(hipMalloc((void**)&census_right_d, width * height * sizeof(uint64_t)));
	gpuErrchk(hipMalloc((void**)&cost_aggr_d, (size_t)width * height * disparity_range * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&cost_init_d, (size_t)width * height * disparity_range * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&cross_arm_left_d, width * height * sizeof(CrossArm)));
	gpuErrchk(hipMalloc((void**)&vec_counter_horizontal_d, width * height * sizeof(uint16_t)));
	gpuErrchk(hipMalloc((void**)&vec_counter_vertical_d, width * height * sizeof(uint16_t)));
	gpuErrchk(hipMalloc((void**)&vec_counter_buffer_d, width * height * sizeof(uint16_t)));
	gpuErrchk(hipMalloc((void**)&disp_mask_d, width * height * sizeof(uint8_t)));
	gpuErrchk(hipMalloc((void**)&disp_left_d, width * height * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&disp_left_buffer_d, width * height * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&disp_right_d, width * height * sizeof(float)));
	
	//set memory to 0 (if neccessary,seems not..)
	gpuErrchk(hipMemset(census_left_d, 0, width * height * sizeof(uint64_t)));
	gpuErrchk(hipMemset(census_right_d, 0, width * height * sizeof(uint64_t)));
	gpuErrchk(hipMemset(cost_aggr_d, 0, (size_t)width * height * disparity_range * sizeof(float)));
	gpuErrchk(hipMemset(cost_init_d, 0, (size_t)width * height * disparity_range * sizeof(float)));
	gpuErrchk(hipMemset(vec_counter_horizontal_d, 0, width * height * sizeof(uint16_t)));
	gpuErrchk(hipMemset(vec_counter_vertical_d, 0, width * height * sizeof(uint16_t)));
	gpuErrchk(hipMemset(vec_counter_buffer_d, 0, width * height * sizeof(uint16_t)));
	gpuErrchk(hipMemset(disp_mask_d, 0, width * height * sizeof(uint8_t)));
	gpuErrchk(hipMemset(disp_left_d, 0, width * height * sizeof(float)));
	gpuErrchk(hipMemset(disp_left_buffer_d, 0, width * height * sizeof(float)));
	gpuErrchk(hipMemset(disp_right_d, 0, width * height * sizeof(float)));
	//transfer memory 


	disp_left_h = (float*)calloc((size_t)width * height, sizeof(float));
	disp_right_h = (float*)calloc((size_t)width * height, sizeof(float));
	//allocate multiple stream
	if (this->speedup_use_multiple_stream) 
	{
		for (int i = 0; i < stream_num; i++)
		{
			gpuErrchk(hipStreamCreate(&streams[i]));
		}
	}

}

void ADCensusStereo::FreeCudaResource()
{
	gpuErrchk(hipFree(img_left_gray_d));
	gpuErrchk(hipFree(img_right_gray_d));
	gpuErrchk(hipFree(img_left_rgb_d));
	gpuErrchk(hipFree(img_right_rgb_d));
	gpuErrchk(hipFree(census_left_d));
	gpuErrchk(hipFree(census_right_d));
	gpuErrchk(hipFree(cost_aggr_d));
	gpuErrchk(hipFree(cost_init_d));
	gpuErrchk(hipFree(cross_arm_left_d));
	gpuErrchk(hipFree(vec_counter_horizontal_d));
	gpuErrchk(hipFree(vec_counter_vertical_d));
	gpuErrchk(hipFree(vec_counter_buffer_d));
	gpuErrchk(hipFree(disp_mask_d));
	gpuErrchk(hipFree(disp_left_d));
	gpuErrchk(hipFree(disp_left_buffer_d));
	gpuErrchk(hipFree(disp_right_d));

	for (int i = 0; i < stream_num; i++)
	{    
	   gpuErrchk(hipStreamDestroy(streams[i]));
	}
	//free(streams);
	free(disp_left_h);
	free(disp_right_h);
}

void ADCensusStereo::Init()
{   
	if (first_time) {
		AllocateCudaResource();
		first_time = false;
	}
	else {
	    //do nothing
	}
}

void ADCensusStereo::Reset()
{
	CleanUpMemory();
}

void ADCensusStereo::SetComputeImg(uint8_t* left_img, uint8_t* right_img)
{
	this->img_left_rgb_h = left_img;
	this->img_right_rgb_h = right_img;
	gpuErrchk(hipMemcpy(img_left_rgb_d, img_left_rgb_h, width * height * 3 * sizeof(uint8_t), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(img_right_rgb_d, img_right_rgb_h, width * height * 3 * sizeof(uint8_t), hipMemcpyHostToDevice));
}

void ADCensusStereo::Compute()
{

	CostCompute(); //ok
	CostAggregate();//ok
	ScanLineOptimize();//ok, ok first two optimize
	MultiStepRefine();
}

float* ADCensusStereo::RetrieveLeftDisparity()
{   
	
	return disp_left_h;
}

float* ADCensusStereo::RetrieveRightDisparity()
{   
	return disp_right_h;
}




void ADCensusStereo::CostCompute()
{   
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	constexpr unsigned int BLOCK_SIZE = 32;
	unsigned int block_dim_x = height / BLOCK_SIZE + 1;
	unsigned int block_dim_y = width / BLOCK_SIZE + 1;
	dim3 blockconfig = { BLOCK_SIZE,BLOCK_SIZE,1 };
	dim3 gridconfig = { block_dim_x,block_dim_y,1 };
	if (this->speedup_use_multiple_stream) {	
		compute_gray <<< gridconfig, blockconfig, 0, streams[0] >>> (img_left_rgb_d, img_left_gray_d, height, width);
		compute_gray <<< gridconfig, blockconfig, 0, streams[1] >>> (img_right_rgb_d, img_right_gray_d, height, width);
		//gpuErrchk(hipPeekAtLastError());
		census_transform_97 <<< gridconfig, blockconfig, 0, streams[0] >>> (img_left_gray_d,census_left_d,height,width);
		census_transform_97 <<< gridconfig, blockconfig, 0, streams[1]>>> (img_right_gray_d,census_right_d,height,width);
		for (int i = 0; i < 2; i++)
			hipStreamSynchronize(streams[i]);
		//gpuErrchk(hipPeekAtLastError());
		
		compute_cost <<< gridconfig, blockconfig, 0, streams[0] >>> (img_left_rgb_d,
			                                                         img_left_gray_d,
			                                                         img_right_rgb_d,
			                                                         img_right_gray_d,
			                                                         cost_init_d,
			                                                         census_left_d,
			                                                         census_right_d,
			                                                         m_option,
			                                                         height,
			                                                         width                                                
		                                                         );
         
		hipStreamSynchronize(streams[0]);
		//gpuErrchk(hipPeekAtLastError());
	}
	else {
		//synchornize using default null stream, no need to config more
		compute_gray <<< gridconfig, blockconfig >>> (img_left_rgb_d, img_left_gray_d, height, width);
		//gpuErrchk(hipPeekAtLastError());
		compute_gray <<< gridconfig, blockconfig >>> (img_right_rgb_d, img_right_gray_d, height, width);
		//gpuErrchk(hipPeekAtLastError());
		census_transform_97 <<< gridconfig, blockconfig >>> (img_left_gray_d, census_left_d, height, width);
		//gpuErrchk(hipPeekAtLastError());
		census_transform_97 <<< gridconfig, blockconfig >>> (img_right_gray_d, census_right_d, height, width);
		//gpuErrchk(hipPeekAtLastError());
		compute_cost <<< gridconfig, blockconfig>>> (img_left_rgb_d,
			img_left_gray_d,
			img_right_rgb_d,
			img_right_gray_d,
			cost_init_d,
			census_left_d,
			census_right_d,
			m_option,
			height,
			width
			);
		hipDeviceSynchronize();
		gpuErrchk(hipPeekAtLastError());
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Ellaped time of cost init:%f ms\n", milliseconds);
}

void ADCensusStereo::CostAggregate()
{   
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	constexpr unsigned int BLOCK_SIZE = 32;
	unsigned int block_dim_x = height / BLOCK_SIZE + 1;
	unsigned int block_dim_y = width / BLOCK_SIZE + 1;
	dim3 blockconfig = { BLOCK_SIZE,BLOCK_SIZE,1 };
	dim3 gridconfig = { block_dim_x,block_dim_y,1 };
	//In case your image is pretty small, you could try to use cooperative group, but you are responsible to caculate the maximum
	// capacity for the GPU
	BuildArm <<< gridconfig, blockconfig,0,streams[0] >>> (img_left_rgb_d,cross_arm_left_d,m_option);
	ComputeSubpixelCountHorizontal <<< gridconfig, blockconfig,0,streams[0]>>> (cross_arm_left_d,vec_counter_buffer_d,m_option);
	ComputeSubpixelAggregateHorizontal <<< gridconfig, blockconfig,0,streams[0]>> > (cross_arm_left_d,vec_counter_horizontal_d,vec_counter_buffer_d,m_option);
	ComputeSubpixelCountVertical <<< gridconfig, blockconfig,0,streams[0]>>> (cross_arm_left_d,vec_counter_buffer_d,m_option);
	ComputeSubpixelAggregateVertical <<< gridconfig, blockconfig,0,streams[0]>> > (cross_arm_left_d, vec_counter_vertical_d, vec_counter_buffer_d, m_option);
	hipStreamSynchronize(streams[0]);
	gpuErrchk(hipPeekAtLastError());
	
	unsigned int disp_range = m_option.max_disparity - m_option.min_disparity;
	dim3 worker_gridconfig = { height,width,1 };
	dim3 worker_blockconfig = { disp_range,1,1 };
    //horizontal_first,then vertical
	///iter1 horizontal
	bool graphCreated = false;
	hipGraph_t graph;
	hipGraphExec_t instance;
	for (int it = 0; it < 2; it++)
	{
		if (!graphCreated) {
			hipStreamBeginCapture(streams[0], hipStreamCaptureMode::hipStreamCaptureModeGlobal);
			AggregateInArms1stphase << <worker_gridconfig, worker_blockconfig, 0, streams[0] >> > (cost_init_d, cost_aggr_d, cross_arm_left_d, true, m_option);
			AggregateInArms2ndphase << <worker_gridconfig, worker_blockconfig, 0, streams[0] >> > (cost_init_d, cost_aggr_d, cross_arm_left_d, vec_counter_horizontal_d, true, m_option);
			///iter2 vertical
			//hipStreamSynchronize(streams[0]);
			AggregateInArms1stphase << <worker_gridconfig, worker_blockconfig, 0, streams[0] >> > (cost_init_d, cost_aggr_d, cross_arm_left_d, false, m_option);
			AggregateInArms2ndphase << <worker_gridconfig, worker_blockconfig, 0, streams[0] >> > (cost_init_d, cost_aggr_d, cross_arm_left_d, vec_counter_vertical_d, false, m_option);
			hipStreamEndCapture(streams[0], &graph);
			hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
			graphCreated = true;
		}
	    hipGraphLaunch(instance, streams[0]);
		hipStreamSynchronize(streams[0]);
	}
	///iter3 horizontal
	//hipStreamSynchronize(streams[0]);
	//AggregateInArms1stphase << <worker_gridconfig, worker_blockconfig, 0, streams[0] >> > (cost_init_d, cost_aggr_d, cross_arm_left_d, true, m_option);
	//AggregateInArms2ndphase << <worker_gridconfig, worker_blockconfig, 0, streams[0] >> > (cost_init_d, cost_aggr_d, cross_arm_left_d, vec_counter_horizontal_d, true, m_option);
	///iter4 vertical
	//hipStreamSynchronize(streams[0]);
	//AggregateInArms1stphase << <worker_gridconfig, worker_blockconfig, 0, streams[0] >> > (cost_init_d, cost_aggr_d, cross_arm_left_d, false, m_option);
	//AggregateInArms2ndphase << <worker_gridconfig, worker_blockconfig, 0, streams[0] >> > (cost_init_d, cost_aggr_d, cross_arm_left_d, vec_counter_vertical_d, false, m_option);
	hipStreamSynchronize(streams[0]); 
	//gpuErrchk(hipPeekAtLastError());
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Ellaped time of aggregation:%f ms\n", milliseconds);
}

void ADCensusStereo::ScanLineOptimize()
{   
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	//after 4iteration the cost of iterative refinement come back to cost_init_d, so start as cost_init_d
	unsigned int disp_range = m_option.max_disparity - m_option.min_disparity;
	dim3 worker_left2rightgridconfig = { height,1,1 };
	dim3 worker_top2bottomgridconfig = { width,1,1 };
	dim3 worker_blockconfig = { disp_range,1,1 };
	unsigned int shared_mem_usage = disp_range * 2*sizeof(float);
	scanline_optimize_left2right<<<worker_left2rightgridconfig, worker_blockconfig, shared_mem_usage, streams[0]>>>(img_left_rgb_d, img_right_rgb_d, cost_init_d, cost_aggr_d, m_option);
	scanline_optimize_right2left<<<worker_left2rightgridconfig, worker_blockconfig, shared_mem_usage, streams[0]>>>(img_left_rgb_d, img_right_rgb_d, cost_aggr_d, cost_init_d, m_option);
	scanline_optimize_top2bottom<<<worker_top2bottomgridconfig, worker_blockconfig, shared_mem_usage, streams[0]>>>(img_left_rgb_d, img_right_rgb_d, cost_init_d, cost_aggr_d, m_option);
	scanline_optimize_bottom2top<<<worker_top2bottomgridconfig, worker_blockconfig, shared_mem_usage, streams[0]>>>(img_left_rgb_d, img_right_rgb_d, cost_aggr_d, cost_init_d, m_option);
	hipStreamSynchronize(streams[0]);
	//gpuErrchk(hipPeekAtLastError());
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Ellaped time of scan line optimize:%f ms\n", milliseconds);
}

void ADCensusStereo::MultiStepRefine()
{   
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	unsigned int disp_range = m_option.max_disparity - m_option.min_disparity;
	unsigned int shared_mem_usage = disp_range * 2 * sizeof(float);
	dim3 worker_gridconfig = { height,width,1 };
	dim3 worker_blockconfig = { disp_range,1,1 };
	retrieve_left_disp <<< worker_gridconfig, worker_blockconfig,shared_mem_usage,streams[0] >>> (cost_init_d,disp_left_d,m_option);
	retrieve_right_disp <<< worker_gridconfig,worker_blockconfig,shared_mem_usage,streams[0] >>> (cost_init_d,disp_right_d, m_option);
	hipStreamSynchronize(streams[0]);
	gpuErrchk(hipPeekAtLastError());
	constexpr unsigned int BLOCK_SIZE = 16;
	unsigned int block_dim_x = height / BLOCK_SIZE + 1;
	unsigned int block_dim_y = width / BLOCK_SIZE + 1;
	dim3 blockconfig = { BLOCK_SIZE,BLOCK_SIZE,1 };
	dim3 gridconfig = { block_dim_x,block_dim_y,1 };
	dim3 interpolationgridconfig = {height,width,1};
	dim3 interpolationblockconfig = { 16,1,1 };
	if (m_option.do_filling) {
		outlier_detection << <gridconfig, blockconfig, 0, streams[0] >> > (disp_left_d, disp_right_d, disp_left_buffer_d, disp_mask_d, m_option);
		//do region voting 4 times
		
		region_voting << < gridconfig, blockconfig, 0, streams[0] >> > (disp_left_buffer_d, disp_left_d, disp_mask_d, cross_arm_left_d, m_option);
		region_voting << < gridconfig, blockconfig, 0, streams[0] >> > (disp_left_d, disp_left_buffer_d, disp_mask_d, cross_arm_left_d, m_option);
		region_voting << < gridconfig, blockconfig, 0, streams[0] >> > (disp_left_buffer_d, disp_left_d, disp_mask_d, cross_arm_left_d, m_option);
		region_voting << < gridconfig, blockconfig, 0, streams[0] >> > (disp_left_d, disp_left_buffer_d, disp_mask_d, cross_arm_left_d, m_option);
		//interpolation << < interpolationgridconfig, interpolationblockconfig, 0, streams[0] >> > (disp_left_buffer_d, disp_left_d, img_left_rgb_d, disp_mask_d, m_option);
		//median_filter << < gridconfig, blockconfig, 0, streams[0] >> > (disp_left_d, disp_left_buffer_d, height, width);
	}
	else {
		outlier_detection << <gridconfig, blockconfig, 0, streams[0] >> > (disp_left_d, disp_right_d, disp_left_buffer_d, disp_mask_d, m_option);
	}
	//median_filter << < gridconfig, blockconfig, 0, streams[0] >> > (disp_left_buffer_d, disp_left_d, height, width);
	hipStreamSynchronize(streams[0]);
	gpuErrchk(hipPeekAtLastError());
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Ellaped time of postprocessing:%f ms\n", milliseconds);
	float* disp_output_d = nullptr;
	disp_output_d = disp_left_buffer_d;
	gpuErrchk(hipMemcpy(disp_left_h, disp_output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(disp_right_h, disp_right_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
	gpuErrchk(hipPeekAtLastError());
	/*
	uint8_t* disp_mask_h = (uint8_t*)malloc(sizeof(uint8_t) * height * width);
	hipMemcpy(disp_mask_h, disp_mask_d, sizeof(uint8_t)*height*width,hipMemcpyDeviceToHost);
	int occlusion_sum = 0;
	int mismatch_sum = 0;
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++)
		{
			uint8_t mask = disp_mask_h[i * width + j];
			if (mask == OCCLUSIONS)
				occlusion_sum += 1;
			if (mask == MISMATCHES)
				mismatch_sum += 1;
		}
	}
	*/
}
